#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
        __global__ void naiveScan(int n, int *odata, const int *idata, int d) {
            int idx = threadIdx.x + (blockIdx.x * blockDim.x);
            if (idx >= n) return;
            if (idx >= (1 << (d - 1))) {
                odata[idx] = idata[idx - (1 << (d - 1))] + idata[idx];
            }
            else {
                odata[idx] = idata[idx];
            }
            
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO

            int* tempIn;
            int* tempOut;
            hipMalloc((void**)&tempIn, n * sizeof(int));
            hipMalloc((void**)&tempOut, n * sizeof(int));
            hipMemset(tempIn, 0, sizeof(int));
            hipMemcpy(tempIn + 1, idata, (n - 1) * sizeof(int), hipMemcpyHostToDevice);
            
            for (int d = 1; d <= ilog2ceil(n); ++d) {
                naiveScan << < 1, std::min(1024, n) >> > (n, tempOut, tempIn, d); // Check that this is the right blocksize!!
                std::swap(tempOut, tempIn);
            }

            std::swap(tempOut, tempIn);

            hipMemcpy(odata, tempOut, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(tempIn);
            hipFree(tempOut);

            timer().endGpuTimer();
        }
    }
}
